#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>

// ����������� �� GPU
__global__
void matmulDevice(int* A, int* B, int* C, int N)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	if (row < N && col < N) {
		int sum = 0;
		for (int i = 0; i < N; i++)
			sum += A[row * N + i] * B[i * N + col];
		C[row * N + col] = sum;
	}
}

void matmulHost(int* A, int* B, int* C, int N)
{

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			int sum = 0;
			for (int k = 0; k < N; k++)
				sum += A[i * N + k] * B[k * N + j];
			C[i * N + j] = sum;
		}
	}
}

#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

using namespace std;

int main(void)
{
	int device_count = 0;
	    hipGetDeviceCount(&device_count);
	
		if (device_count == 0)
			cout << "Sorry! You dont have CudaDevice" << endl;
		else
			cout << "CudaDevice found! Device count: " << device_count << endl;
	
		int N = 2048;
		int block_size = 16;
		// ���-�� ��������
		int nIter = 1;

		unsigned int count = N*N;
		unsigned int mem_size = sizeof(int) * count;
	
	
		int* A = (int*)malloc(mem_size);
		int* B = (int*)malloc(mem_size);
		int* h_C = (int*)malloc(mem_size);
		int* hCuda_C = (int*)malloc(mem_size);
	
		int* d_A, * d_B, * d_C;
	
		for (int i = 0; i < count; i++) {
			A[i] = rand() % 100 + 1;
			B[i] = rand() % 100 + 1;
		}
	
		unsigned int start_time = clock();

		for (int j = 0; j < nIter; j++) {
			matmulHost(A, B, h_C, N);
		}

		unsigned int elapsedTime = clock() - start_time;
		float msecPerMatrixMulCpu = elapsedTime / nIter;

		cout << "CPU time: " << msecPerMatrixMulCpu << endl;
	
		checkCudaErrors(hipMalloc((void**)& d_A, mem_size));
		checkCudaErrors(hipMalloc((void**)& d_B, mem_size));
		checkCudaErrors(hipMalloc((void**)& d_C, mem_size));
	
		// �������� ������ �� ������
		checkCudaErrors(hipMemcpy(d_A, A, mem_size,
			hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_B, B, mem_size,
			hipMemcpyHostToDevice));
	
		dim3 threadsPerBlock(block_size, block_size);
		dim3 blocksPerGrid(N / block_size, N / block_size);
		
		hipEvent_t start;
		hipEvent_t stop;
		checkCudaErrors(hipEventCreate(&start));
		checkCudaErrors(hipEventCreate(&stop));

		// ���������� ������ �������
		checkCudaErrors(hipEventRecord(start, 0));

		for (int j = 0; j < nIter; j++) {
			matmulDevice << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, N);
		}

		// ���������� ����� �������
		checkCudaErrors(hipEventRecord(stop, 0));

		// ���� ����� �������
		checkCudaErrors(hipEventSynchronize(stop));

		float msecTotal = 0.0f;
		checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

		float msecPerMatrixMul = msecTotal / nIter;
			   
		cout << "GPU time: " << msecPerMatrixMul << endl;

		hipDeviceSynchronize();
	
		// �������� ��������� � �������
		checkCudaErrors(hipMemcpy(hCuda_C, d_C, mem_size, hipMemcpyDeviceToHost));
		hipDeviceSynchronize();
	
		// free device memory
		hipFree(d_A);
	    hipFree(d_B);
	    hipFree(d_C);
	
		bool test = true;
	
		for (int i = 0; i < count; i++) {
			if (h_C[i] != hCuda_C[i])
				test = false;
		}

		if (test)
			cout << "PASS!" << endl;
		else 
			cout << "WASTED!" << endl;
	
    return 0;
}

